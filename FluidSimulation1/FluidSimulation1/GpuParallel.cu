#include "hip/hip_runtime.h"
#include "GpuParallel.cuh"

float smoothingKernel(float radius, float distance) {
	if (distance >= radius) {
		return 0.0f;
	}

	float x = (radius - distance) / radius;
	return x * x;
}

__global__ void updateParticleDensitiesKernel(Particle* particles, InteractionMatrixClass* interactionMatrix, int particleRadiusOfRepel) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	Particle* particle = &particles[index];

	Vector2D point = particle->m_PredictedPosition;

	std::vector<Particle*> particlesInCell = interactionMatrix->getParticlesInCell(point, particleRadiusOfRepel);

	constexpr auto scalar = 1000;

	float density = 0.0f;
	const float mass = 1.0f;

	for (int i = 0; i < particlesInCell.size(); i++) {
		float distance = sqrt(Math::squared_distance(point, particle->m_PredictedPosition));
		float influence = smoothingKernel(particleRadiusOfRepel, distance);
		density += mass * influence;
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	density = density / volume * scalar;

	particle->m_Density = density;
}



void GpuParallelUpdateParticleDensities(std::vector<Particle*> particles, InteractionMatrixClass* interactionMatrix, int particleRadiusOfRepel) {


	std::vector<Particle*>* cudaParticles;
	InteractionMatrixClass* cudaInteractionMatrix;
	int* cudaParticleRadiusOfRepel;

	hipMalloc(&cudaParticles, particles.size() * sizeof(Particle*));
	hipMalloc(&cudaInteractionMatrix, sizeof(InteractionMatrixClass));
	hipMalloc(&cudaParticleRadiusOfRepel, sizeof(int));

	hipMemcpy(cudaParticles, particles.data(), particles.size() * sizeof(Particle*), hipMemcpyHostToDevice);
	hipMemcpy(cudaInteractionMatrix, interactionMatrix, sizeof(InteractionMatrixClass), hipMemcpyHostToDevice);
	hipMemcpy(cudaParticleRadiusOfRepel, &particleRadiusOfRepel, sizeof(int), hipMemcpyHostToDevice);

	updateParticleDensitiesKernel << <1, particles.size() >> > (cudaParticles, cudaInteractionMatrix, *cudaParticleRadiusOfRepel);

	int x = 0;
}